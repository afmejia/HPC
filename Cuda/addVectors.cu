#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

const int size = 1000;

void fillVectors(float *A);
void printVector(float *V);
void sumVectors(float *A, float *B, float *C);
//void vecAddKernel(float *A, float *B, float *C);

int main(int argc, char const *argv[]) {
  float *A = (float *) malloc(size * sizeof(float));
  float *B = (float *) malloc(size * sizeof(float));
  float *C = (float *) malloc(size * sizeof(float));
//  int *C;
//  B = (int *) malloc(size);
//  C = (int *) malloc(size);
  fillVectors(A);
  fillVectors(B);
  sumVectors(A, B, C);
  printVector(A);
  printVector(B);
  printVector(C);
  free(A);
  free(B);
  free(C);
  return 0;
}


void fillVectors(float *A) {
  for (int i = 0; i < size; i++)  {
    A[i] = i + 1;
  }
}

void printVector(float *V) {
  for (int i = 0; i < size; i++)
    printf("%d ", (int)V[i]);
  printf("\n");
}

__global__
void vecAddKernel(float *A, float *B, float *C)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < size)
    C[i] = A[i] + B[i];
}

void sumVectors(float *A, float *B, float *C) {
  int n = size * sizeof(float);
  float *d_A, *d_B, *d_C;

  // Allocate device memory for A, B, and C
  // copy A and B to device memory
  hipMalloc((void**)&d_A, n);
  hipMemcpy(d_A, A, n, hipMemcpyHostToDevice);
  hipMalloc((void**)&d_B, n);
  hipMemcpy(d_B, B, n, hipMemcpyHostToDevice);
  hipMalloc((void**)&d_C, n);

  // Kernel launch code - to have the device to perform the actual vector addition
  // Run ceil(size/256) blocks of 256 threads each
  vecAddKernel <<< ceil(size/256.0), 256 >>> (d_A, d_B, d_C);

  // copy C from the device memory
  hipMemcpy(C, d_C, n, hipMemcpyDeviceToHost);

  // Free device vectors
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
