#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WIDTH 2

void fillMatrix(float *A, int A_height, int A_width);
void printMatrix(float *A, int A_height, int A_width);
void tiledMat(float *A, float *result, int A_width, int A_height);

int main(int argc, char const *argv[])
{
        // Create matrix
        int A_width = 4;
        int A_height = 4;
        float *A = (float*) malloc(A_width * A_height * sizeof(float));
        float *result = (float*) malloc(A_width * A_height * sizeof(float));
        fillMatrix(A, A_height, A_width);
        printMatrix(A, A_height, A_width);
        tiledMat(A, result, A_width, A_height);
        printMatrix(result, A_height, A_width);
        free(A);
        return 0;
}

__global__
void blockTranspose(float *A_elements, int A_width, int A_height)
{
        __shared__ float blockA[BLOCK_WIDTH][BLOCK_WIDTH];
        int BLOCK_SIZE = BLOCK_WIDTH;
        int baseIdx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
        baseIdx += (blockIdx.y * BLOCK_SIZE + threadIdx.y) * A_width;
        blockA[threadIdx.y][threadIdx.x] = A_elements[baseIdx];
        A_elements[baseIdx] = blockA[threadIdx.x][threadIdx.y];
}

void fillMatrix(float *A, int A_height, int A_width)
{
        int size = A_height * A_width;
        for(int i = 0; i < size; i++)
                A[i] = (float) i + 1;
}

void printMatrix(float *A, int A_height, int A_width)
{
        for (int i = 0; i < A_height; i++)
        {
                for (int j = 0; j < A_width; j++)
                {
                        printf("%d\t", (int)A[i * A_width + j]);
                }
                printf("\n");
        }
}

void tiledMat(float *A, float *result, int A_width, int A_height)
{
        // Create device matrix
        float *d_A;
        int A_size = A_width * A_height * sizeof(float);
        hipError_t err = hipMalloc((void **) &d_A, A_size);
        if (err != hipSuccess)
        {
                printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
                exit(EXIT_FAILURE);
        }
        hipMemcpy(d_A, A, A_size, hipMemcpyHostToDevice);

        // Launch the kernel
        dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH);
        dim3 griDim(A_width / blockDim.x, A_height / blockDim.y);
        blockTranspose<<<griDim, blockDim>>>(d_A, A_width, A_height);
        hipMemcpy(result, d_A, A_size, hipMemcpyDeviceToHost);
        hipFree(d_A);
}
