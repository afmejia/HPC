#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WIDTH 3

void fillMatrix(float *A, int A_height, int A_width);
void printMatrix(float *A, int A_height, int A_width);
void tileddMat(int A_width, int A_height);

int main(int argc, char const *argv[])
{
        // Create matrix
        int A_width = 9;
        int A_height = 9;
        float *A = (float*) malloc(A_width * A_height * sizeof(float));
        fillMatrix(A, A_height, A_with);
        tiledMat(A_with, A_height);
        return 0;
}

void fillMatrix(float *A, int A_height, int A_width)
{
  int size = A_height * A_width;
  for(int i = 0; i < size; i++)
    A[i] = (float) i + 1;
}

void printMatrix(float *A, int A_height, int A_width)
{
  int size = A_height * A_width;
  for (int i = 0; i < A_height; i++)
  {
    for (int j = 0; j < A_width; j++)
    {
      printf("%d\t", (int)A[i * A_width + j]);
    }
    printf("\n");
  }
}

void tiledMat(int A_width, int A_height)
{
        dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH);
        dim3 griDim(A_width / blockDim.x, A_height / blockDim.y);
}
