#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>

using namespace cv;
using namespace std;

Mat& filter(Mat& image);
Mat& gpuFilter(Mat& image);

int main(int argc, char const *argv[]) {
        // Load the image
        if (argc != 2)
        {
                cout << "Usage: ./twoFilter.out <Image_Path>" << endl;
                return -1;
        }

        String imageName = argv[1];
        Mat image;
        image = imread(imageName, IMREAD_COLOR);

        if (image.empty())
        {
                cout << "Could not open or find the image." << endl;
                return -1;
        }

        // Apply filter
        Mat result = image.clone();
        result = gpuFilter(result);

        //Show image
        /*namedWindow("Original image", WINDOW_AUTOSIZE);
        imshow("landscape", image);
        //namedWindow("Filtered image", WINDOW_AUTOSIZE);
        imshow("filtered landscape", result);
        waitKey(0);*/
        return 0;
}

Mat& filter(Mat& image)
{
        // Accept only char type matrices
        CV_Assert(image.depth() == CV_8U);

        // Create iterator and iterate over the whole image
        MatIterator_<Vec3b> it, end;

        for (it = image.begin<Vec3b>(), end = image.end<Vec3b>(); it != end; ++it)
        {
                (*it)[0] = (*it)[0] * 2;
                (*it)[1] = (*it)[1] * 2;
                (*it)[2] = (*it)[2] * 2;
        }

        return image;
}

Mat& gpuFilter(Mat& image)
{
        // Accept only char type matrices
        CV_Assert(image.depth() == CV_8U);

        // Define image size in the device memory
        int channels = image.channels();
        int cols = image.cols * channels;
        int rows = image.rows;
        int im_size = cols * rows * sizeof(uchar);

        // Flat host image
        uchar* h_img;
        h_img = img.ptr<uchar>(0);

        // Create device image
        uchar* d_img;

        // Allocate device memory for the image
        hipError_t err = hipMalloc((void **) &d_img, im_size);
        if (err != hipSuccess)
        {
          cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__;
          exit(EXIT_FAILURE);
        }

        // Copy image from host to device
        hipMemcpy(d_img, h_img, im_size, hipMemcpyHostToDevice);

        cout << "Success" << endl;
        hipFree(d_img);



        // Create iterator and iterate over the whole image
        MatIterator_<Vec3b> it, end;

        for (it = image.begin<Vec3b>(), end = image.end<Vec3b>(); it != end; ++it)
        {
                (*it)[0] = (*it)[0] * 2;
                (*it)[1] = (*it)[1] * 2;
                (*it)[2] = (*it)[2] * 2;
        }

        return image;
}

/*__global__ void PictureKernell(float* d_Pin, float* d_Pout, int n, int m)
   {
   // Calculate the row # of the d_Pin and d_Pout element to process
   int Row = blockIdx.y * blockDim.y + threadIdx.y;

   // Calculate the column # of the d_Pin and d_Pout element to process
   int Col = blockIdx.x * blockDim.x + threadIdx.x;

   //each thread computes one element of d_Pout if in range
   if ((Row < m) && (Col < n))
    d_Pout[Row * n + Col] = 2 d_Pin[Row * n + Col];
   }*/

