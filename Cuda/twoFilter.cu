#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace cv;
using namespace std;

Mat& filter(Mat& image);
Mat gpuFilter(Mat& image);

int main(int argc, char const *argv[]) {
        // Load the image
        if (argc != 2)
        {
                cout << "Usage: ./twoFilter.out <Image_Path>" << endl;
                return -1;
        }

        String imageName = argv[1];
        Mat image;
        image = imread(imageName, IMREAD_COLOR);

        if (image.empty())
        {
                cout << "Could not open or find the image." << endl;
                return -1;
        }

        // Apply filter
        Mat result;
        result = gpuFilter(image);

        //Show image
        namedWindow("Original image", WINDOW_AUTOSIZE);
        imshow("landscape", image);
        //namedWindow("Filtered image", WINDOW_AUTOSIZE);
        imshow("filtered landscape", result);
        waitKey(0);
        return 0;
}

__global__ void pictureKernel(uchar* d_img_in, uchar* d_img_out, int rows, int cols)
{
        // Calculate the row # of the d_img element to process
        int row = blockIdx.y * blockDim.y + threadIdx.y;

        // Calculate the column # of the d_img element to process
        int col = blockIdx.x * blockDim.x + threadIdx.x;

        // Each thread computes one element of d_img if in range
        if ((row < rows) && (col < cols))
                d_img_out[row * cols + col] = 2 * d_img_in[row * cols + col];
}

Mat& filter(Mat& image)
{
        // Accept only char type matrices
        CV_Assert(image.depth() == CV_8U);

        // Create iterator and iterate over the whole image
        MatIterator_<Vec3b> it, end;

        for (it = image.begin<Vec3b>(), end = image.end<Vec3b>(); it != end; ++it)
        {
                (*it)[0] = (*it)[0] * 2;
                (*it)[1] = (*it)[1] * 2;
                (*it)[2] = (*it)[2] * 2;
        }

        return image;
}

Mat gpuFilter(Mat& image)
{
        // Accept only char type matrices
        CV_Assert(image.depth() == CV_8U);

        // Define image size in the device memory
        int channels = image.channels();
        int cols = image.cols * channels;
        int rows = image.rows;
        int im_size = cols * rows * sizeof(uchar);

        // Flat host image
        uchar* h_img = (uchar*) image.data;

        // Create device images and result host image
        uchar* d_img_in;
        uchar* d_img_out;
        uchar* h_result = (uchar*) malloc(im_size);

        // Allocate device memory for the images
        hipError_t err = hipMalloc((void **) &d_img_in, im_size);
        if (err != hipSuccess)
        {
                cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__;
                exit(EXIT_FAILURE);
        }

        err = hipMalloc((void**) &d_img_out, im_size);
        if (err != hipSuccess)
        {
                cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__;
                exit(EXIT_FAILURE);
        }

        // Copy image from host to device
        err = hipMemcpy(d_img_in, h_img, im_size, hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
                cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__;
                exit(EXIT_FAILURE);
        }

        // Launch the Kernel
        dim3 dimGrid(ceil(cols / 256.0), ceil(rows / 256.0), 1);
        dim3 dimBlock(16, 16, 1);
        pictureKernel<<<dimGrid, dimBlock>>>(d_img_in, d_img_out, rows, cols);

        // Copy result image from device to host
        err = hipMemcpy(h_result, d_img_out, im_size, hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
                cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__;
                exit(EXIT_FAILURE);
        }
        Mat result(rows, cols, CV_8UC3, (void*) h_result);

        cout << "Success" << endl;
        hipFree(d_img_in);
        hipFree(d_img_out);
        return result;
}

