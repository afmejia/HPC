#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>

using namespace cv;
using namespace std;

Mat& filter(Mat& image);
Mat& gpuFilter(Mat& image, uchar* h_img, uchar* h_imgOut, Size size, int im_size);

int main(int argc, char const *argv[]) {
        // Load the image
        if (argc != 2)
        {
                cout << "Usage: ./twoFilter.out <Image_Path>" << endl;
                return -1;
        }

        String imageName = argv[1];
        Mat image;
        image = imread(imageName, IMREAD_COLOR);

        if (image.empty())
        {
                cout << "Could not open or find the image." << endl;
                return -1;
        }

        // Define image size in host memory
        Size size = image.size();
        int channels = image.channels();
        int width = size.width;
        int height = size.height;
        int im_size = width * height * channels * sizeof(uchar);

        //Create host image container
        uchar* h_img = (uchar*) malloc(im_size);
        uchar* h_imgOut = (uchar*) malloc(im_size);

        // Apply filter
        Mat result = image.clone();
        //resultCpu = filter(result);
        Mat resultGpu = gpuFilter(result, h_img, h_imgOut, size, im_size);

        //Show image
        imshow("landscape", image);
        imshow("filtered landscape", resultGpu);
        waitKey(0);
        free(h_img);
        free(h_imgOut);
        return 0;
}

<<<<<<< HEAD
__global__ void pictureKernel(uchar* d_img_in, uchar* d_img_out, int rows, int cols)
{
=======
/*__global__ void pictureKernel(uchar* d_img_in, uchar* d_img_out, int rows, int cols)
   {
>>>>>>> 9c34741928fa05cd2d29c6608c860598aa0796ee
        // Calculate the row # of the d_img element to process
        int row = blockIdx.y * blockDim.y + threadIdx.y;

        // Calculate the column # of the d_img element to process
        int col = blockIdx.x * blockDim.x + threadIdx.x;

        // Each thread computes one element of d_img if in range
        if ((row < rows) && (col < cols))
                d_img_out[row * cols + col] = 2 * d_img_in[row * cols + col];
<<<<<<< HEAD
}
=======
   }*/
>>>>>>> 9c34741928fa05cd2d29c6608c860598aa0796ee

Mat& filter(Mat& image)
{
        // Accept only char type matrices
        CV_Assert(image.depth() == CV_8U);

        // Create iterator and iterate over the whole image
        MatIterator_<Vec3b> it, end;

        for (it = image.begin<Vec3b>(), end = image.end<Vec3b>(); it != end; ++it)
        {
                (*it)[0] = -(*it)[0] - 2;
                (*it)[1] = -(*it)[1] - 2;
                (*it)[2] = -(*it)[2] - 2;
        }

        return image;
}

Mat& gpuFilter(Mat& image, uchar* h_img, uchar* h_imgOut, Size size, int im_size)
{
        // Accept only char type matrices
        CV_Assert(image.depth() == CV_8U);

        // Create host image
        h_img = image.data;
<<<<<<< HEAD

        // Sequencial filter
        /*for(int i = 0; i < im_size; i++)
           {
           h_imgOut[i] = 2 * h_img[i];
           }
           image.release();
           image.create(size, CV_8UC3);
           image.data = h_imgOut;*/

        // Allocate device memory for the image
        // Copy image to the device
        uchar *d_img, *d_imgOut;
        hipError_t err = hipMalloc((void**) &d_img, im_size);
        if (err != hipSuccess)
        {
                printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
                exit(EXIT_FAILURE);
        }
        hipMemcpy(d_img, h_img, im_size, hipMemcpyHostToDevice);

        // Create image in the device for the result image
        err = hipMalloc((void**) &d_imgOut, im_size);
        if (err != hipSuccess)
        {
                printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
                exit(EXIT_FAILURE);
        }

        //Kernel launch code
        int cols = size.width;
        int rows = size.height;
        dim3 dimGrid(ceil(cols / 16.0), ceil(rows / 16.0), 1);
        dim3 dimBlock(16, 16, 1);
        pictureKernel<<<dimBlock, dimGrid>>>(d_img, d_imgOut, rows, cols);

        // Copy result into the host from the device memory
        hipMemcpy(h_imgOut, d_imgOut, im_size, hipMemcpyDeviceToHost);

        // Put the host image in a Mat container
        image.release();
        image.create(size, CV_8UC3);
        image.data = h_imgOut;
        Mat result(rows, cols / 3, CV_8UC3, (void*)h_img);

        // Free memory
        hipFree(d_img);
        hipFree(d_imgOut);

=======
        //Mat result;
        //result.create(size, CV_8UC3);
        //result.data = h_imgOut;

        // Sequencial filter
        for(int i = 0; i < im_size; i++)
        {
                h_img[i] = 2 * h_img[i];
        }

        // Allocate device memory for the image
        // Copy image to the device
        /*uchar* d_img, d_imgOut;
           hipError_t err = hipMalloc((void**) &d_img, im_size);
           if (err != hipSuccess)
           {
                printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
                exit(EXIT_FAILURE);
           }
           hipMemcpy(d_img, h_img, im_size, hipMemcpyHostToDevice);

           // Create image in the device for the result image
           err = hipMalloc((void**) &d_imgOut, im_size);
           if (err != hipSuccess)
           {
                printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
                exit(EXIT_FAILURE);
           }*/

        //Kernel launch code
        /*int cols = size.width;
           int rows = size.height;
           dim3 dimGrid(ceil(cols / 16.0), ceil(rows / 16.0), 1);
           dim3 dimBlock(16, 16, 1);
           pictureKernel<<<dimBlock, dimGrid>>>(d_img, d_imgOut, rows, cols);

           // Copy result into the host from the device memory
           hipMemcpy(h_img, d_imgOut, im_size, hipMemcpyDeviceToHost);

           // Put the host image in a Mat container
           Mat result(rows, cols / 3, CV_8UC3, (void*)h_img);

           // Free memory
           hipFree(d_img);
           hipFree(d_imgOut);*/

>>>>>>> 9c34741928fa05cd2d29c6608c860598aa0796ee
        return image;
}
