#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace cv;
using namespace std;

Mat& filter(Mat& image);
Mat gpuFilter(Mat& image);

int main(int argc, char const *argv[]) {
        // Load the image
        if (argc != 2)
        {
                cout << "Usage: ./twoFilter.out <Image_Path>" << endl;
                return -1;
        }

        String imageName = argv[1];
        Mat image;
        image = imread(imageName, IMREAD_COLOR);

        if (image.empty())
        {
                cout << "Could not open or find the image." << endl;
                return -1;
        }

        // Apply filter
        Mat result = image.clone();
        //resultCpu = filter(result);
        Mat resultGpu = gpuFilter(result);

        //Show image
        imshow("landscape", image);
	imshow("filtered landscape", resultGpu);
        waitKey(0);
        return 0;
}

/*__global__ void pictureKernel(uchar* d_img_in, uchar* d_img_out, int rows, int cols)
{
        // Calculate the row # of the d_img element to process
        int row = blockIdx.y * blockDim.y + threadIdx.y;

        // Calculate the column # of the d_img element to process
        int col = blockIdx.x * blockDim.x + threadIdx.x;

        // Each thread computes one element of d_img if in range
        if ((row < rows) && (col < cols))
                d_img_out[row * cols + col] = 2 * d_img_in[row * cols + col];
}*/

Mat& filter(Mat& image)
{
        // Accept only char type matrices
        CV_Assert(image.depth() == CV_8U);

        // Create iterator and iterate over the whole image
        MatIterator_<Vec3b> it, end;

        for (it = image.begin<Vec3b>(), end = image.end<Vec3b>(); it != end; ++it)
        {
                (*it)[0] = - (*it)[0] - 2;
                (*it)[1] = - (*it)[1] - 2;
                (*it)[2] = - (*it)[2] - 2;
        }

        return image;
}

Mat gpuFilter(Mat& image)
{
        // Accept only char type matrices
        CV_Assert(image.depth() == CV_8U);

        // Define image size in host memory
	Size size = image.size();
	int channels = image.channels();
	int width = size.width;
	int height = size.height;
	int im_size = width * height * channels * sizeof(uchar);


	// Create host image
	uchar* h_img = (uchar*) malloc(im_size);
	h_img = image.data;
	Mat result;
	result.create(size, CV_8UC3);
	result.data = h_img;

	// Sequencial filter
	for(int i = 0; i < im_size; i++)
	{
		h_img[i] = 2 * h_img[i];
	}
	
	//TODO: Finally the fucking kernel with the two filter
	
        // Flat host image
        //uchar* h_img = (uchar*) image.data;
	//Mat result(rows, cols, CV_8UC3, (void*) h_img);

	// Sequencial filter using sizeof
	/*int size = sizeof(h_img);
	for(int i = 0; i < size; i++)
	{
		h_img[i] = h_img[i] * 2;
	}*/

	//Mat result(rows, cols, CV_8UC3, (void*) h_img);

        /*// Create device images and result host image
        uchar* d_img_in;
        uchar* d_img_out;
        uchar* h_result = (uchar*) malloc(im_size);

        // Allocate device memory for the images
        hipError_t err = hipMalloc((void **) &d_img_in, im_size);
        if (err != hipSuccess)
        {
                cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__;
                exit(EXIT_FAILURE);
        }

        err = hipMalloc((void**) &d_img_out, im_size);
        if (err != hipSuccess)
        {
                cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__;
                exit(EXIT_FAILURE);
        }

        // Copy image from host to device
        err = hipMemcpy(d_img_in, h_img, im_size, hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
                cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__;
                exit(EXIT_FAILURE);
        }

        // Launch the Kernel
        dim3 dimGrid(ceil(cols / 256.0), ceil(rows / 256.0), 1);
        dim3 dimBlock(16, 16, 1);
        pictureKernel<<<dimGrid, dimBlock>>>(d_img_in, d_img_out, rows, cols);

        // Copy result image from device to host
        err = hipMemcpy(h_result, d_img_out, im_size, hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
                cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__;
                exit(EXIT_FAILURE);
        }
        Mat result(rows, cols, CV_8UC3, (void*) h_result);

        cout << "Success" << endl;
        hipFree(d_img_in);
        hipFree(d_img_out);*/
        return result;
}


