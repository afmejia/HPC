#include "hip/hip_runtime.h"
__global__ void PictureKernell(float* d_Pin, float* d_Pout, int n, int m)
{
  // Calculate the row # of the d_Pin and d_Pout element to process
  int Row = blockIdx.y * blockDim.y + threadIdx.y;

  // Calculate the column # of the d_Pin and d_Pout element to process
  int Col = blockIdx.x * blockDim.x + threadIdx.x;

  //each thread computes one element of d_Pout if in range
  if ((Row < m) && (Col < n))
    d_Pout[Row * n + Col] = 2 d_Pin[Row * n + Col];
}

