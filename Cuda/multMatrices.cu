
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

const long int M = 10;
const long int N = 10;
const long int O = 10;
#define BLOCK_WIDTH 16

void fillMatrix(float *A, long long int sizeA);
void printMatrix(float *A, long long int sizeA, long long int N);
float getValue(float *A, long long int i, long long int j, long long int cols);
void setValue(float *A, long long int i, long long int j, float value);
void multiMatricesCPU(float *A, float *B, float *C);
void multiMatricesGPU(float *A, float *B, float *C);

int main(int argc, char const *argv[])
{
        //Creting the sizes
        long long int sizeA = M * N;
        long long int sizeB = M * O;
        long long int sizeC = M * O;

        //Creating the matrices
        float A[sizeA];
        float B[sizeB];
        float C[sizeC];

        //Filling the matrices with secuencial numbers
        fillMatrix(A, sizeA);
        fillMatrix(B, sizeB);
        fillMatrix(C, sizeC);

        // Multiplying A and B
        //multiMatricesCPU(A, B, C);
        multiMatricesGPU(A, B, C);
        //printMatrix(A, sizeA, N);
        printMatrix(C, sizeC, O);
        return 0;
}

__global__ void matrixMultKernel (float *d_A, float *d_B, float *d_C, int N)
{
  // Calculate the row index of the d_C element and d_A
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  // Calculate the column index of d_C and d_B
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if ((row < N) && (col < N))
  {
    float Cvalue = 0;
    for (int k = 0; k < N; k++)
      Cvalue += d_A[row * N + k] * d_B[k * N + col];
    d_C[row * N + col] = Cvalue;
  }
}

void fillMatrix(float *A, long long int sizeA)
{
        for (long long int i = 0; i < sizeA; i++)
                A[i] = i+1;
}

// Get the value in the position (i, j) in matrix A
float getValue(float *A, long long int i, long long int j, long long int cols)
{
        return A[cols * i + j];
}

// Set the value on the position (i, j) of matrix A
void setValue(float *A, long long int i, long long int j, float value)
{
        A[O * i + j] = value;
}

// Print the entire matrix A
void printMatrix(float *A, long long int sizeA, long long int N)
{
        for (long long int i = 0; i < sizeA; i++)
        {
                if (i % N == 0 && i != 0)
                        printf("\n");
                printf("%d ", (int) A[i]);
        }
        printf("\n");
}

// Computes the multiplications between matrices A and B and stores the result on Matrix C
void multiMatricesCPU(float *A, float *B, float *C)
{
        long long int temp = 0;
        for (long long int i = 0; i < M; i++)
        {
                for (long long int j = 0; j < O; j++)
                {
                        for (long long int k = 0; k < N; k++)
                        {
                                temp = temp + getValue(A, i, k, N) * getValue(B, k, j, O);
                        }
                        setValue(C, i, j, temp);
                        temp = 0;
                }
        }
}

void multiMatricesGPU(float *h_A, float *h_B, float *h_C)
{
        // Define sizes of matrices in device memory
        int A_size = N * N * sizeof(float);
        int B_size = A_size;
        int C_size = A_size;

        // Create device matrices
        float *d_A, *d_B, *d_C;

        // Allocate device memory for matrices
        // copy host matrices to device matrices
        hipError_t err = hipMalloc((void **) &d_A, A_size);
        if (err != hipSuccess)
        {
                printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
                exit(EXIT_FAILURE);
        }
        hipMemcpy(d_A, h_A, A_size, hipMemcpyHostToDevice);

        err = hipMalloc((void **) &d_B, B_size);
        if (err != hipSuccess)
        {
                printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
                exit(EXIT_FAILURE);
        }
        hipMemcpy(d_B, h_B, B_size, hipMemcpyHostToDevice);

        err = hipMalloc((void **) &d_C, C_size);
        if (err != hipSuccess)
        {
                printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
                exit(EXIT_FAILURE);
        }

        // Launch kernel
        int NumBlocks = N / BLOCK_WIDTH;
        if (N % BLOCK_WIDTH)
                NumBlocks++;
        dim3 dimGrid(NumBlocks, NumBlocks);
        dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
        matrixMultKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

        // Put results again on host conrainers
        hipMemcpy(h_C, d_C, C_size, hipMemcpyDeviceToHost);
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
}

