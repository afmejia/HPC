
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

const long int M = 10;
const long int N = 10;
const long int O = 10;

void fillMatrix(long long int *A, long long int sizeA);
void printMatrix(long long int *A, long long int sizeA, long long int N);
long long int getValue(long long int *A, long long int i, long long int j, long long int cols);
void setValue(long long int *A, long long int i, long long int j, long long int value);
void multiMatricesCPU(long long int *A, long long int *B, long long int *C);
void multiMatricesGPU(long long int *A, long long int *B, long long int *C);

int main(int argc, char const *argv[])
{
        //Creting the sizes
        long long int sizeA = M * N;
        long long int sizeB = M * O;
        long long int sizeC = M * O;

        //Creating the matrices
        long long int A[sizeA];
        long long int B[sizeB];
        long long int C[sizeC];

        //Filling the matrices with secuencial numbers
        fillMatrix(A, sizeA);
        fillMatrix(B, sizeB);
        fillMatrix(C, sizeC);

        // Multiplying A and B
        multiMatricesCPU(A, B, C);
        //printMatrix(A, sizeA, N);
        //printMatrix(C, sizeC, O);
        return 0;
}

// TODO write the kernel

void fillMatrix(long long int *A, long long int sizeA)
{
        for (long long int i = 0; i < sizeA; i++)
                A[i] = i+1;
}

// Get the value in the position (i, j) in matrix A
long long int getValue(long long int *A, long long int i, long long int j, long long int cols)
{
        return A[cols * i + j];
}

// Set the value on the position (i, j) of matrix A
void setValue(long long int *A, long long int i, long long int j, long long int value)
{
        A[O * i + j] = value;
}

// Print the entire matrix A
void printMatrix(long long int *A, long long int sizeA, long long int N)
{
        for (long long int i = 0; i < sizeA; i++)
        {
                if (i % N == 0 && i != 0)
                        printf("\n");
                printf("%lld ", A[i]);
        }
        printf("\n");
}

// Computes the multiplications between matrices A and B and stores the result on Matrix C
void multiMatricesCPU(long long int *A, long long int *B, long long int *C)
{
        long long int temp = 0;
        for (long long int i = 0; i < M; i++)
        {
                for (long long int j = 0; j < O; j++)
                {
                        for (long long int k = 0; k < N; k++)
                        {
                                temp = temp + getValue(A, i, k, N) * getValue(B, k, j, O);
                        }
                        setValue(C, i, j, temp);
                        temp = 0;
                }
        }
}

void multiMatricesGPU(float *h_A, float *h_B, float *h_C)
{
        // Define sizes of matrices in device memory
        int A_size = N * N * sizeof(float);
        int B_size = A_size;
        int C_size = A_size;

        // Create device matrices
        float *d_A, *d_B, *d_C;

        // Allocate device memory for matrices
        // copy host matrices to device matrices
        hipError_t err = hipMalloc((void **) &d_A, A_size);
        if (err != hipSuccess)
        {
                printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
                exit(EXIT_FAILURE);
        }
        hipMemcpy(d_A, h_A, A_size, hipMemcpyHostToDevice);

        err = hipMalloc((void **) &d_B, B_size);
        if (err != hipSuccess)
        {
                printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
                exit(EXIT_FAILURE);
        }
        hipMemcpy(d_B, h_B, B_size, hipMemcpyHostToDevice);

        err = hipMalloc((void **) &d_C, C_size);
        if (err != hipSuccess)
        {
                printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
                exit(EXIT_FAILURE);
        }

        // Launch kernel
}

