#include <stdio.h>

int main(int argc, char const *argv[]) {
        int dev_count;
        hipGetDeviceCount(&dev_count);
        printf("There are %d cuda Devices\n", dev_count);
        hipDeviceProp_t dev_prop;
        for (int i = 0; i < dev_count; i++)
        {
                hipGetDeviceProperties(&dev_prop, i);
                printf("Device %d: \n", i);
                printf("Maximum number of threads per block: %d \n", dev_prop.maxThreadsPerBlock);
                printf("Number of SMs in the device: %d \n", dev_prop.multiProcessorCount);
                printf("Clockrate: %d \n", dev_prop.clockRate);
                printf("Maximum threads in x = %d, y = %d and z = %d \n",
                       dev_prop.maxThreadsDim[0], dev_prop.maxThreadsDim[1],
                       dev_prop.maxThreadsDim[2]);
                printf("Maximum size of grid in x = %d, y = %d, z = %d \n",
                       dev_prop.maxGridSize[0], dev_prop.maxGridSize[1],
                       dev_prop.maxGridSize[2]);
                printf("\n");
        }
        return 0;
}

