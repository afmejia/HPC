
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char const *argv[]) {
  int dev_count;
  hipGetDeviceCount(&dev_count);
  printf("There are %d cuda Devices\n", dev_count);
  return 0;
}
