#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10

void addMatrices(float *h_A, float *h_B, float *h_C);
void fillMatrix(float *A);
void printMatrix(float *A);

int main(int argc, char const *argv[]) {
  float *A = (float *) malloc(N * N * sizeof(float));
  float *B = (float *) malloc(N * N * sizeof(float));
  float *C = (float *) malloc(N * N * sizeof(float));
  fillMatrix(A);
  fillMatrix(B);
  addMatrices(A, B, C);
  printMatrix(C);
  free(A);
  free(B);
  free(C);
  return 0;
}

void fillMatrix(float *A)
{
  int size = N * N;
  for (int i = 0; i < size; i++)
  {
    A[i] = i + 1;
  }
}

void addMatrices(float *h_A, float *h_B, float *h_C)
{
    int size = N * N;
    int d_size = N * N * sizeof(float);}
    float *d_A, *d_B, *d_C;

    // Allocate device memory for A, B, and C
    // copy h_A and h_B to device memory
    hipError_t err = hipMalloc((void**) &d_A, size);
    if (err != hipSuccess)
    {
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
      exit(EXIT_FAILURE);
    }
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMalloc((void**) &d_B, size);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipMalloc((void**) &d_C, size);

    // Kernel launch code - to have the device to perform the actual matrix addition

    // copy C from the device memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device vector (which represents our matrices)
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    for (int i = 0; i < size; i++)
    {
      h_C[i] = h_A[i] + h_B[i];
    }
}

void printMatrix(float *A)
{
  int size = N * N;
  for (int i = 0; i < size; i++)
  {
    if (i % N == 0 && i != 0)
      printf("\n");
    printf("%d\t", (int)A[i]);
  }
}
