#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 1000

void addMatrices(float *h_A, float *h_B, float *h_C);
void fillMatrix(float *h_A);
void printMatrix(float *A);

int main(int argc, char const *argv[]) {
  float *h_A = (float *) malloc(N * N * sizeof(float));
  float *h_B = (float *) malloc(N * N * sizeof(float));
  float *h_C = (float *) malloc(N * N * sizeof(float));
  fillMatrix(h_A);
  fillMatrix(h_B);
  addMatrices(h_A, h_B, h_C);
  printMatrix(h_C);
  free(h_A);
  free(h_B);
  free(h_C);
  return 0;
}

__global__
void matAddKernel(float *d_A, float *d_B, float *d_C, int size)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int element;
  if (i < size)
  {
    for (int j = 0; j < size; j++)
    {
      element = i * size + j;
      d_C[element] = d_A[element] + d_B[element];
      //printf("Element %d from thread %d\n", element, i);
    }
  }
}

void fillMatrix(float *h_A)
{
  int size = N * N;
  for (int i = 0; i < size; i++)
  {
    h_A[i] = i + 1;
  }
}

void addMatrices(float *h_A, float *h_B, float *h_C)
{
    int size = N * N;
    int d_size = size * sizeof(float);
    float *d_A, *d_B, *d_C;

    // Allocate device memory for A, B, and C
    // copy h_A and h_B to device memory
    hipError_t err = hipMalloc((void**) &d_A, d_size);
    if (err != hipSuccess)
    {
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
      exit(EXIT_FAILURE);
    }
    hipMemcpy(d_A, h_A, d_size, hipMemcpyHostToDevice);

    err = hipMalloc((void**) &d_B, d_size);
    if (err != hipSuccess)
    {
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
      exit(EXIT_FAILURE);
    }
    hipMemcpy(d_B, h_B, d_size, hipMemcpyHostToDevice);

    err = hipMalloc((void**) &d_C, d_size);
    if (err != hipSuccess)
    {
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
      exit(EXIT_FAILURE);
    }

    // Kernel launch code - to have the device to perform the actual matrix addition
    matAddKernel<<<ceil((N)/256.0), 256>>>(d_A, d_B, d_C, N);

    // copy C from the device memory
    hipMemcpy(h_C, d_C, d_size, hipMemcpyDeviceToHost);

    // Free device vector (which represents our matrices)

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void printMatrix(float *A)
{
  int size = N * N;
  for (int i = 0; i < size; i++)
  {
    if (i % N == 0 && i != 0)
      printf("\n");
    printf("%d\t", (int)A[i]);
  }
}

