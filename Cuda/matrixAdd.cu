#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10

void addMatrices(float *h_A, float *h_B, float *h_C);
void fillMatrix(float *A);
void printMatrix(float *A);

int main(int argc, char const *argv[]) {
  float *A = (float *) malloc(N * N * sizeof(float));
  float *B = (float *) malloc(N * N * sizeof(float));
  float *C = (float *) malloc(N * N * sizeof(float));
  fillMatrix(A);
  fillMatrix(B);
  addMatrices(A, B, C);
  //printMatrix(A);
  //printMatrix(B);
  printMatrix(C);
  free(A);
  free(B);
  free(C);
  return 0;
}

__global__
void matAddKernel(float *A, float *B, float *C, int size)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < size)
    C[i] = A[i] + B[i];
}

void fillMatrix(float *A)
{
  int size = N * N;
  for (int i = 0; i < size; i++)
  {
    A[i] = i + 1;
  }
}

void addMatrices(float *h_A, float *h_B, float *h_C)
{
    int size = N * N;
    int d_size = N * N * sizeof(float);
    float *d_A, *d_B, *d_C;

    // Allocate device memory for A, B, and C
    // copy h_A and h_B to device memory
    hipMalloc((void**)&d_A, d_size);
    hipMalloc((void**)&d_B, d_size);
    hipMalloc((void**)&d_C, d_size);
    /*cudaError_t err = cudaMalloc((void**) &d_A, size);
    if (err != cudaSuccess)
    {
      printf("%s in %s at line %d\n", cudaGetErrorString(err), __FILE__, __LINE__);
      exit(EXIT_FAILURE);
    }*/
    hipMemcpy(d_A, h_A, d_size, hipMemcpyHostToDevice);

    /*err = cudaMalloc((void**) &d_B, size);
    if (err != cudaSuccess)
    {
      printf("%s in %s at line %d\n", cudaGetErrorString(err), __FILE__, __LINE__);
      exit(EXIT_FAILURE);
    }*/
    hipMemcpy(d_B, h_B, d_size, hipMemcpyHostToDevice);

    /*err = cudaMalloc((void**) &d_C, d_size);
    if (err != cudaSuccess)
    {
      printf("%s in %s at line %d\n", cudaGetErrorString(err), __FILE__, __LINE__);
      exit(EXIT_FAILURE);
    }*/

    // Kernel launch code - to have the device to perform the actual matrix addition
    matAddKernel<<<ceil(size/256.0), 256>>>(d_A, d_B, d_C, size);

    // copy C from the device memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device vector (which represents our matrices)
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void printMatrix(float *A)
{
  int size = N * N;
  for (int i = 0; i < size; i++)
  {
    if (i % N == 0 && i != 0)
      printf("\n");
    printf("%d\t", (int)A[i]);
  }
  printf("\n");
}

