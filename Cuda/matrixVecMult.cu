#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 5

void fillMatrix(float *h_A, int size);
void mult(float *h_B, float *h_C, float *h_A, int n);
void printVector(float *h_A, int size);

int main(int argc, char const *argv[]) {
  // Input matrix h_B and input vector h_C
  float *h_B = (float *) malloc(N * N * sizeof(float));
  float *h_C = (float *) malloc(N * sizeof(float));

  // Result vector h_A
  float *h_A = (float *) malloc(N * sizeof(float));

  // Fill vector h_C and matrix h_B
  fillMatrix(h_C, N);
  fillMatrix(h_B, N * N);

  // Save dot product between h_B and h_C in h_A
  mult(h_B, h_C, h_A, N);

  // Print the result vector
  printVector(h_A, N);
  return 0;
}

void fillMatrix(float *h_A, int size)
{
  for (int i = 0; i < size; i++)
    h_A[i] = i + 1;
}

void mult(float *h_B, float *h_C, float *h_A, int n)
{
  // Define sizes of matrix and vectors in device memory
  int B_size = N * N * sizeof(float);
  int C_size = N * sizeof(float);

  // Create device arrays
  float *d_A, *d_B, *d_C;

  // Allocate device memory for A, B, and C
  // copy h_B and h_C to device memory
  hipError_t err = hipMalloc((void **) &d_B, B_size);
  if (err != hipSuccess)
  {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
  hipMemcpy(d_B, h_B, B_size, hipMemcpyHostToDevice);

  err = hipMalloc((void **) &d_C, c_Size);
  if (err != hipSuccess)
  {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
  hipMemcpy(d_C, h_C, c_size, hipMemcpyHostToDevice);

  err = hipMalloc((void **) &d_A, C_size);
  if (err != hipSuccess)
  {
    printf("%s in %s at line %d", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }

  // TODO
  // Launch kernel for each row
  // Test everything


  float temp = 0;
  int j = 0;
  for (int i = 0; i < n; i++)
  {
    for (int k = 0; k < n; k++)
    {
      temp = temp + h_B[j] * h_C[k];
      j++;
    }
    h_A[i] = temp;
    temp = 0;
  }
}

void printVector(float *h_A, int size)
{
  for (int i = 0; i < size; i++)
    printf("%d ", (int)h_A[i]);
  printf("\n");
}
