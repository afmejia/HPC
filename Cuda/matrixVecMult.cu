#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define N 5

void fillMatrix(float *h_A, int size);
void mult(float *h_B, float *h_C, float *h_A, int n);
void printVector(float *h_A, int size);

int main(int argc, char const *argv[]) {
  // Input matrix h_B and input vector h_C
  float *h_B = (float *) malloc(N * N * sizeof(float));
  float *h_C = (float *) malloc(N * sizeof(float));

  // Result vector h_A
  float *h_A = (float *) malloc(N * sizeof(float));

  // Fill vector h_C and matrix h_B
  fillMatrix(h_C, N);
  fillMatrix(h_B, N * N);

  // Save dot product between h_B and h_C in h_A
  mult(h_B, h_C, h_A, N);

  // Print the result vector
  printVector(h_A, N);

  //Free host memory
  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}

__global__
void dotProduct(float *d_B, float *d_C, float *d_A, int n)
{
  float temp = 0;
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < n)
  {
    for (int j = 0; j < n; j++)
      temp = temp + (d_B[i * n + j] * d_C[j]);
    d_A[i] = temp;
  }
}

void fillMatrix(float *h_A, int size)
{
  for (int i = 0; i < size; i++)
    h_A[i] = i + 1;
}

void mult(float *h_B, float *h_C, float *h_A, int n)
{
  // Define sizes of matrix and vectors in device memory
  int B_size = N * N * sizeof(float);
  int C_size = N * sizeof(float);

  // Create device arrays
  float *d_A, *d_B, *d_C;

  // Allocate device memory for A, B, and C
  // copy h_B and h_C to device memory
  hipError_t err = hipMalloc((void **) &d_B, B_size);
  if (err != hipSuccess)
  {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
  hipMemcpy(d_B, h_B, B_size, hipMemcpyHostToDevice);

  err = hipMalloc((void **) &d_C, C_size);
  if (err != hipSuccess)
  {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
  hipMemcpy(d_C, h_C, C_size, hipMemcpyHostToDevice);

  err = hipMalloc((void **) &d_A, C_size);
  if (err != hipSuccess)
  {
    printf("%s in %s at line %d", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }

  // Launch kernel for each row
  dotProduct<<<ceil(n / 256.0), 256>>>(d_B, d_C, d_A, n);

  // Copy the result vector from devive to host
  hipMemcpy(h_A, d_A, C_size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_A);
  hipFree(d_C);
  hipFree(d_B);
}

void printVector(float *h_A, int size)
{
  for (int i = 0; i < size; i++)
    printf("%d ", (int)h_A[i]);
  printf("\n");
}

